
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
#include <string>
#include <vector>
#include <thread>

__global__ void helloCUDA()
{
    printf("Hello, CUDA!\n");
}

void runRT() {

}

bool cuda_avail() {
    int devCount;
    hipGetDeviceCount(&devCount);
    return devCount > 0;
}


void cuda_hello()
{
    int devCount;
    hipGetDeviceCount(&devCount);
    for(int i = 0; i < devCount; ++i)
    {
        hipDeviceProp_t props;
        hipGetDeviceProperties(&props, i);
        printf("%d: %s:%d.%d\n", i, props.name, props.major, props.minor);
    }
    helloCUDA<<<3, 1>>>();
    hipDeviceSynchronize();
    return;
}